#include <cstdlib>
#include <iostream>
#include <ctime>
#include <utility>  //std::pair
#include <cmath>
#include <thrust/host_vector.h>
#include <chrono>

#include "k_means_cpu.hpp"
#include "k_means_gpu_1.cuh"
#include "k_means_gpu_2.cuh"

#include "utils.h"

using namespace std;


int main()
{
    unsigned seed = (unsigned)time(NULL);
    srand(seed);
    cout << "Seed for srand: " << seed << endl;
    const int N = 200000, n = 3, k = 60;
    const double threshold = 1e-4;
    auto input_pair = generate_sample_input(N, n, k);
    cout << "\nN=" << N << " , n=" << n << " , k=" << k << " , threshold=" << threshold << endl;
    pair<thrust::host_vector<double>, thrust::host_vector<int>> cpu_result, gpu_result1, gpu_result2;
    cout << "\nk_means_gpu_1::Compute:\n";
    START_STOPWATCH
    gpu_result1 = k_means_gpu_1::Compute(input_pair.second.begin(), N, n, k, threshold);
    STOP_STOPWATCH
    cout << "\nk_means_gpu_2::Compute:\n";
    START_STOPWATCH
    gpu_result2 = k_means_gpu_2::Compute(input_pair.second.begin(), N, n, k, threshold);
    STOP_STOPWATCH
    cout << "\nk_means_cpu::Compute:\n";
    START_STOPWATCH
    cpu_result = k_means_cpu::Compute(input_pair.first.begin(), N, n, k, threshold);
    STOP_STOPWATCH
    cout << endl;
	cout << boolalpha << "Cpu result with Gpu1 comparison: " << compare_cpu_gpu_results(cpu_result, gpu_result1, n, k)
	    << endl;
    cout << boolalpha << "Gpu1 result with Gpu2 result comparison: " << compare_gpu_results(gpu_result1, gpu_result2)
        << endl;
    return 0;
}
